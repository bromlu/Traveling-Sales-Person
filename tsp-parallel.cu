#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <getopt.h>
#include <limits.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <limits.h>
#include <math.h>

#define ONE_BILLION (double)1000000000.0
#define TSP_ELT(tsp, n, i, j) *(tsp + (i * n) + j)

/* Print a TSP distance matrix. */
void
print_tsp(int *tsp, int n, int random_seed)
{
printf("TSP (%d cities - seed %d)\n    ", n, random_seed);
for (int j = 0;  j < n;  j++) {
printf("%3d|", j);
}
printf("\n");
for (int i = 0;  i < n;  i++) {
printf("%2d|", i);
for (int j = 0;  j < n;  j++) {
printf("%4d", TSP_ELT(tsp, n, i, j));
}
printf("\n");
}
printf("\n");
}

/* Print a permutation array */
__host__ __device__ void
print_perm(int *perm, int size)
{
  printf("Min Path: ");
  for (int k = 0; k < size; k++) {
	printf("%4d", perm[k]);
  }
  printf("\n");
}

/**** List ADT ****************/

typedef struct {
    int *values;					/* Values stored in list */
    int max_size;					/* Maximum size allocated */
    int cur_size;					/* Size currently in use */
} list_t;

/* Dump list, including sizes */
__host__ __device__ void
list_dump(list_t *list)
{
    printf("%2d/%2d", list->cur_size, list->max_size);
    for (int i = 0;  i < list->cur_size;  i++) {
        printf(" %d", list->values[i]);
    }
    printf("\n");
}

/* Allocate list that can store up to 'max_size' elements */
__host__ __device__ list_t *
list_alloc(int max_size)
{
    list_t *list = (list_t *)malloc(sizeof(list_t));
    list->values = (int *)malloc(max_size * sizeof(int));
    list->max_size = max_size;
    list->cur_size = 0;
    return list;
}

/* Free a list; call this to avoid leaking memory! */
__host__ __device__ void
list_free(list_t *list)
{
    free(list->values);
    free(list);
}

/* Add a value to the end of the list */
__host__ __device__ void
list_add(list_t *list, int value)
{
    if (list->cur_size >= list->max_size) {
        printf("List full");
        list_dump(list);
    } else {
        list->values[list->cur_size++] = value;
    }
}

/* Return the current size of the list */
__host__ __device__ int
list_size(list_t *list)
{
    return list->cur_size;
}

/* Validate index */
__host__ __device__ void
_list_check_index(list_t *list, int index)
{
    if (index < 0 || index > list->cur_size - 1) {
        printf("Invalid index %d\n", index);
        list_dump(list);
    }
}

/* Get the value at given index */
__host__ __device__ int
list_get(list_t *list, int index)
{
    _list_check_index(list, index);
    return list->values[index];
}

/* Remove the value at the given index */
__host__ __device__ void
list_remove_at(list_t *list, int index)
{
    _list_check_index(list, index);
    for (int i = index; i < list->cur_size - 1;  i++) {
        list->values[i] = list->values[i + 1];
    }
    list->cur_size--;
}

/* Retrieve a copy of the values as a simple array of integers. The returned
array is allocated dynamically; the caller must free the space when no
longer needed.
*/
__host__ __device__ int *
list_as_array(list_t *list)
{
    int *rtn = (int *)malloc(list->max_size * sizeof(int));
    for (int i = 0;  i < list->max_size;  i++) {
        rtn[i] = list_get(list, i);
    }
    return rtn;
}

/* Calculate n! iteratively */
__host__ __device__ unsigned long
factorial(int n)
{
    if (n < 1) {
        return 0;
    }

    unsigned long rtn = 1;
    for (int i = 1;  i <= n;  i++) {
        rtn *= i;
    }
    return rtn;
}

/* Return the kth lexographically ordered permuation of an array of k integers
   in the range [0 .. size - 1]. The integers are allocated dynamically and
   should be free'd by the caller when no longer needed.
*/
__host__ __device__ int *
kth_perm(unsigned long k, int size)
{
    unsigned long remain = k;

    list_t *numbers = list_alloc(size);
    for (int i = 0;  i < size;  i++) {
        list_add(numbers, i);
    }

    list_t *perm = list_alloc(size);

    for (int i = 1;  i < size;  i++) {
        unsigned long f = factorial(size - i);
        unsigned long j = remain / f;
        remain = remain % f;

        list_add(perm, list_get(numbers, j));
        list_remove_at(numbers, j);

        if (remain == 0) {
            break;
        }
    }

    /* Append remaining digits */
    for (int i = 0;  i < list_size(numbers);  i++) {
        list_add(perm, list_get(numbers, i));
    }

    int *rtn = list_as_array(perm);
    list_free(perm);

    return rtn;
}

/* Swap v[i] and v[j] */
__device__ void
swap(int *v, int i, int j)
{
    int t = v[i];
    v[i] = v[j];
    v[j] = t;
}

/* Given an array of size elements at perm, update the array in place to
   contain the lexographically next permutation. It is originally due to
   Dijkstra. The present version is discussed at:
   http://www.cut-the-knot.org/do_you_know/AllPerm.shtml
 */
__device__ void
next_perm(int *perm, int size)
{
    int i = size - 1;
    while (perm[i - 1] >= perm[i]) {
        i = i - 1;
    }

    int j = size;
    while (perm[j - 1] <= perm[i - 1]) {
        j = j - 1;
    }

    swap(perm, i - 1, j - 1);

    i++;
    j = size;
    while (i < j) {
        swap(perm, i - 1, j - 1);
        i++;
        j--;
    }
}

__device__ int 
calc_cost(int* tsp, int* perm, int num_cities) {
    int total = 0;
    for (int i = 0;  i < num_cities;  i++) {
        int j = (i + 1) % num_cities;
        int from = perm[i];
        int to = perm[j];
        int val = TSP_ELT(tsp, num_cities, from, to);
        total += val;
    }
    return total;
}

/* TSP Kernal */
__global__ void
TSP(int* tsp, int* mins, unsigned long* min_perms, unsigned long total_permutations, unsigned long num_threads, unsigned long num_cities) {
    int idx = threadIdx.x;
    unsigned long permutations_per_thread = total_permutations / num_threads;
    unsigned long perm_idx = idx * permutations_per_thread;
    unsigned long stop_idx = (idx + 1) * permutations_per_thread;
    if(idx == num_threads - 1) {
        stop_idx = total_permutations;
    }

    // printf("Thread %d | permutations_per_thread: %ld | perm_idx: %ld | stop_idx: %ld\n", idx, permutations_per_thread, perm_idx, stop_idx);

    int * perm = kth_perm(perm_idx, num_cities);
    int min = INT_MAX;
    unsigned long min_perm = perm_idx;
    
    while(perm_idx < stop_idx) {
        // printf("Thread %d | min: %d\n", idx, min);
        printf("Thread %d | permutations_per_thread: %ld | perm_idx: %ld | stop_idx: %ld\n", idx, permutations_per_thread, perm_idx, stop_idx);
        int cost = calc_cost(tsp, perm, num_cities);
        if(cost < min) {
            min = cost;
            min_perm = perm_idx;
        }
        next_perm(perm, num_cities);
        perm_idx = perm_idx + 1;
    }

    mins[idx] = min;
    min_perms[idx] = min_perm;
}

/* Create an instance of a symmetric TSP. */
int *
create_tsp(int size, int random_seed, int tsp_size)
{
    int *tsp = (int *)malloc(tsp_size);

    srandom(random_seed);
    for (int i = 0;  i < size;  i++) {
        for (int j = 0;  j <= i;  j++) {
            int val = (int)(random() / (RAND_MAX / 100));
            TSP_ELT(tsp, size, i, j) = val;
            TSP_ELT(tsp, size, j, i) = val;
        }
    }
    return tsp;
}

/* Return the current time. */
double now(void)
{
  struct timespec current_time;
  clock_gettime(CLOCK_REALTIME, &current_time);
  return current_time.tv_sec + (current_time.tv_nsec / ONE_BILLION);
}

/* Print out help */
void
usage(char *prog_name)
{
  fprintf(stderr, "usage: %s [flags]\n", prog_name);
  fprintf(stderr, "   -h\n");
  fprintf(stderr, "   -t <number of threads>\n");
  fprintf(stderr, "   -c <number of cities>\n");
  fprintf(stderr, "   -s <random seed>\n");
  exit(1);
}

int 
main(int argc, char **argv) {

    int random_seed = time(NULL);
    int num_threads = 0;
    int num_cities = 0;

    int ch;
    while ((ch = getopt(argc, argv, "c:hs:t:")) != -1) {
        switch (ch) {
            case 'c':
                num_cities = atoi(optarg);
            break;
            case 't':
                num_threads = atoi(optarg);
            break;
            case 's':
                random_seed = atoi(optarg);
            break;
            case 'h':
            default:
                usage(argv[0]);
            }
    }

    if(num_cities <= 0 || num_threads <= 0) {
        fprintf(stderr, "Error, number of cities and threads must be above 0\n");
        usage(argv[0]);
    }

    double start_time = now();
    
    // Copy tsp cost map to GPU global memory
    int tsp_size = num_cities * num_cities * sizeof(int);
    int mins_size = num_threads * sizeof(int);
    int min_perms_size = num_threads * sizeof(unsigned long);

    int* h_tsp = create_tsp(num_cities, random_seed, tsp_size);
    int* h_mins = (int*)malloc(mins_size);
    unsigned long* h_min_perms = (unsigned long*)malloc(min_perms_size);

    int* d_tsp;
    int* d_mins;
    unsigned long* d_min_perms;

    hipMalloc((void **)&d_tsp, tsp_size);
    hipMalloc((void **)&d_mins, mins_size);
    hipMalloc((void **)&d_min_perms, min_perms_size);

    hipMemcpy(d_tsp, h_tsp, tsp_size, hipMemcpyHostToDevice);

    unsigned long total_permutations = factorial(num_cities);
    fprintf(stderr, "fact: %ld\n", total_permutations);
    // Call the kernel.
    TSP<<<1, num_threads>>>(d_tsp, d_mins, d_min_perms, total_permutations, (unsigned long)num_threads, (unsigned long)num_cities);
    // cudaDeviceSynchronize();

    hipMemcpy(h_mins, d_mins, mins_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_min_perms, d_min_perms, min_perms_size, hipMemcpyDeviceToHost);

    int min = h_mins[0];
    printf("Min Path Cost for %d: %d\n",0, h_mins[0]);
    for(int i = 1; i < num_threads; i++) {
        printf("Min Path Cost for %d: %d\n",i, h_mins[i]);
        if(h_mins[i] < min) {
            min = h_mins[i];
        }
    }

    printf("Min Path Cost: %d\n", min);

    for(int i = 1; i < num_threads; i++) {
        if(h_mins[i] == min) {
            print_perm(kth_perm(h_min_perms[i], num_cities), num_cities);
        }
    }

    /* Report time. */
    printf("    TOOK %5.3f seconds\n", now() - start_time);
}